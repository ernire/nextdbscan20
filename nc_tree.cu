//
// Created by Ernir Erlingsson on 19.8.2020.
//

#include <iostream>
#include <unordered_map>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>
#include <thrust/binary_search.h>
#include "magma_util.h"
#include "nc_tree.h"

struct pack {
    template<typename Tuple>
    __device__ __host__ int64_t operator()(const Tuple &t) {
        return( static_cast<int64_t>( thrust::get<0>(t) ) << 32 ) | thrust::get<1>(t);
    }
};

struct unpack {
    __device__ __host__  thrust::tuple<int,int> operator()(int64_t p) {
        int32_t d = static_cast<int32_t>(p >> 32);
        int32_t s = static_cast<int32_t>(p & 0xffffffff);
        return thrust::make_tuple(d, s);
    }
};

void print_cuda_memory_usage() {
    size_t free_byte;
    size_t total_byte;
    auto cuda_status = hipMemGetInfo( &free_byte, &total_byte );

    if ( hipSuccess != cuda_status ) {
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }
    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

void nc_tree::determine_data_bounds() noexcept {
    v_coord_id.resize(n_coord);
    thrust::sequence(v_coord_id.begin(), v_coord_id.end(), 0);
    v_min_bounds.resize(n_dim);
    v_max_bounds.resize(n_dim);
    thrust::counting_iterator<int> it_cnt_begin(0);
    thrust::counting_iterator<int> it_cnt_end = it_cnt_begin + n_coord;
    for (int d = 0; d < n_dim; ++d) {
        auto it_trans_begin = thrust::make_transform_iterator(it_cnt_begin, (thrust::placeholders::_1 * n_dim) + d);
        auto it_trans_end = thrust::make_transform_iterator(it_cnt_end, (thrust::placeholders::_1 * n_dim) + d);
        auto it_perm_begin = thrust::make_permutation_iterator(v_coord.begin(), it_trans_begin);
        auto it_perm_end = thrust::make_permutation_iterator(v_coord.end(), it_trans_end);
        auto minmax = thrust::minmax_element(it_perm_begin, it_perm_end);
        v_min_bounds[d] = *minmax.first;
        v_max_bounds[d] = *minmax.second;
    }
    v_dim_order.resize(n_dim);
    thrust::sequence(v_dim_order.begin(), v_dim_order.end(), 0);
    auto const i_min_begin = v_min_bounds.begin();
    auto const i_max_begin = v_max_bounds.begin();
    thrust::sort(v_dim_order.begin(), v_dim_order.end(), [=]__device__(int const &d1, int const &d2) -> bool {
        return (*(i_max_begin+d1) - *(i_min_begin+d1)) > (*(i_max_begin+d2) - *(i_min_begin+d2));
    });
}

void nc_tree::index_points(d_vec<float> &v_data, d_vec<int> &v_index) noexcept {
    thrust::counting_iterator<int> it_cnt_begin(0);
    thrust::counting_iterator<int> it_cnt_end = it_cnt_begin + v_index.size();
    auto const it_coords = v_data.begin();
    auto const dim_0 = v_dim_order[0];
    auto const dim_1 = v_dim_order[1];
    float const bound_0 = v_min_bounds[dim_0];
    float const bound_1 = v_min_bounds[dim_1];
    int const mult = v_dim_part_size[0];
    int const dim = n_dim;
    float const ee = e;
    thrust::transform(it_cnt_begin, it_cnt_end, v_index.begin(), [=]__device__(int const &i) -> int {
        return (int)( ( *(it_coords + (i * dim + dim_0)) - bound_0 ) / ee )
               + (int)( ( *(it_coords + (i * dim + dim_1)) - bound_1 ) / ee ) * mult;
    });
}

void nc_tree::initialize_cells() noexcept {
    v_dim_part_size.resize(2);
    v_dim_part_size[0] = (v_max_bounds[v_dim_order[0]] - v_min_bounds[v_dim_order[0]]) / e + 1;
    v_dim_part_size[1] = (v_max_bounds[v_dim_order[1]] - v_min_bounds[v_dim_order[1]]) / e + 1;
//    magma_util::print_v("v_dim_part_size: ", &v_dim_part_size[0], v_dim_part_size.size());
    if (static_cast<uint64_t>(v_dim_part_size[0]) * v_dim_part_size[1] > INT32_MAX) {
        std::cerr << "FAIL: The epsilon value is too low and therefore not supported by the current version for the"
                     " input dataset" << std::endl;
        exit(EXIT_FAILURE);
    }
    v_coord_cell_index.resize(v_coord_id.size());
    d_vec<int> v_point_cell_index(v_coord_id.size());
    index_points(v_coord, v_point_cell_index);

    thrust::sort_by_key(v_point_cell_index.begin(), v_point_cell_index.end(), v_coord_id.begin());
    thrust::counting_iterator<int> it_cnt_begin(0);
    thrust::counting_iterator<int> it_cnt_end = it_cnt_begin + n_coord;
    v_coord_cell_offset.resize(n_coord);
    v_coord_cell_offset[0] = 0;
    auto const it_index = v_point_cell_index.begin();
    auto it = thrust::copy_if(it_cnt_begin + 1, it_cnt_end, v_coord_cell_offset.begin() + 1, [=]__device__(int const &i) -> bool {
       return *(it_index + i - 1) != *(it_index + i);
    });
    v_coord_cell_offset.resize(thrust::distance(v_coord_cell_offset.begin(), it));
    v_coord_cell_index.resize(v_coord_cell_offset.size());
    v_coord_cell_size.resize(v_coord_cell_offset.size());
    auto const it_offset = v_coord_cell_offset.begin();
    it_cnt_end = it_cnt_begin + v_coord_cell_offset.size();
    thrust::transform(it_cnt_begin, it_cnt_end - 1, v_coord_cell_size.begin(), [=]__device__(int const &i) -> int {
        return *(it_offset+i+1) - *(it_offset+i);
    });
    v_coord_cell_size[v_coord_cell_size.size()-1] = n_coord - v_coord_cell_offset[v_coord_cell_size.size()-1];
    thrust::transform(it_cnt_begin, it_cnt_end, v_coord_cell_index.begin(), [=]__device__(int const &i) -> int {
        return *(it_index + *(it_offset + i));
    });
//    std::cout << "v_coord_cell_index begin: " <<  v_coord_cell_index[0] << ", " << v_coord_cell_index[1] << ", " << v_coord_cell_index[2] << std::endl;
//    std::cout << "v_coord_cell_index end: " <<  v_coord_cell_index[v_coord_cell_index.size()-1] << ", " <<
//        v_coord_cell_index[v_coord_cell_index.size()-2] << ", " << v_coord_cell_index[v_coord_cell_index.size()-3] << std::endl;
}

void nc_tree::collect_cells_in_reach(d_vec<int> &v_point_index, d_vec<int> &v_cell_reach,
        d_vec<int> &v_point_reach_offset, d_vec<int> &v_point_reach_size) noexcept {
    int const n_points = v_point_index.size();
    int const dim_part = v_dim_part_size[0];
    d_vec<int> v_point_reach_full(9 * n_points, -1);

    thrust::counting_iterator<int> it_cnt_begin(0);
    thrust::counting_iterator<int> it_cnt_end = it_cnt_begin + n_points;

    d_vec<int> v_lower_bound(n_points * 3);
    auto it_perm_begin = thrust::make_permutation_iterator(v_point_index.begin(), it_cnt_begin);
    auto it_perm_end = thrust::make_permutation_iterator(v_point_index.end(), it_cnt_end);
    auto it_trans_begin_1 = thrust::make_transform_iterator(it_perm_begin, thrust::placeholders::_1 - 1);
    auto it_trans_end_1 = thrust::make_transform_iterator(it_perm_end, thrust::placeholders::_1 - 1);
    auto it_trans_out_1 = thrust::make_transform_iterator(it_cnt_begin, thrust::placeholders::_1 * 3);
    auto it_perm_out_1 = thrust::make_permutation_iterator(v_lower_bound.begin(), it_trans_out_1);
    thrust::lower_bound(v_coord_cell_index.begin(), v_coord_cell_index.end(), it_trans_begin_1, it_trans_end_1, it_perm_out_1);

    auto it_trans_begin_2 = thrust::make_transform_iterator(it_perm_begin, thrust::placeholders::_1 - dim_part - 1);
    auto it_trans_end_2 = thrust::make_transform_iterator(it_perm_end, thrust::placeholders::_1 - dim_part - 1);
    auto it_trans_out_2 = thrust::make_transform_iterator(it_cnt_begin, (thrust::placeholders::_1 * 3) + 1);
    auto it_perm_out_2 = thrust::make_permutation_iterator(v_lower_bound.begin(), it_trans_out_2);
    thrust::lower_bound(v_coord_cell_index.begin(), v_coord_cell_index.end(), it_trans_begin_2, it_trans_end_2, it_perm_out_2);

    auto it_trans_begin_3 = thrust::make_transform_iterator(it_perm_begin, thrust::placeholders::_1 + dim_part - 1);
    auto it_trans_end_3 = thrust::make_transform_iterator(it_perm_end, thrust::placeholders::_1 + dim_part - 1);
    auto it_trans_out_3 = thrust::make_transform_iterator(it_cnt_begin, (thrust::placeholders::_1 * 3) + 2);
    auto it_perm_out_3 = thrust::make_permutation_iterator(v_lower_bound.begin(), it_trans_out_3);
    thrust::lower_bound(v_coord_cell_index.begin(), v_coord_cell_index.end(), it_trans_begin_3, it_trans_end_3, it_perm_out_3);

    auto const it_coord_index = v_coord_cell_index.begin();
    auto const it_value = it_perm_begin;
    auto const it_bounds = v_lower_bound.begin();
    auto const it_full_reach = v_point_reach_full.begin();
    auto const it_reach_size = v_point_reach_size.begin();
    thrust::for_each(it_cnt_begin, it_cnt_end, [=]__device__(int const &i) -> void {

        // TODO RETURN < 0 !!

        auto it_begin = it_full_reach + (i * 9);
        auto it_out = it_begin;
        int val = *(it_bounds + (i * 3));
        if (*(it_coord_index + val) == *(it_value + i) - 1) {
            *(it_out++) = val;
            ++val;
        }
        if (*(it_coord_index + val) == *(it_value + i)) {
            *(it_out++) = val;
            ++val;
        }
        if (*(it_coord_index + val) == *(it_value + i) + 1) {
            *(it_out++) = val;
            ++val;
        }
        val = *(it_bounds + (i * 3) + 1);
        if (*(it_coord_index + val) == *(it_value + i) - dim_part - 1) {
            *(it_out++) = val;
            ++val;
        }
        if (*(it_coord_index + val) == *(it_value + i) - dim_part) {
            *(it_out++) = val;
            ++val;
        }
        if (*(it_coord_index + val) == *(it_value + i) - dim_part + 1) {
            *(it_out++) = val;
            ++val;
        }
        val = *(it_bounds + (i * 3) + 2);
        if (*(it_coord_index + val) == *(it_value + i) + dim_part - 1) {
            *(it_out++) = val;
            ++val;
        }
        if (*(it_coord_index + val) == *(it_value + i) + dim_part) {
            *(it_out++) = val;
            ++val;
        }
        if (*(it_coord_index + val) == *(it_value + i) + dim_part + 1) {
            *(it_out++) = val;
            ++val;
        }
        *(it_reach_size + i) = it_out - it_begin;
    });
    v_cell_reach.resize(v_point_reach_full.size());
    thrust::exclusive_scan(v_point_reach_size.begin(), v_point_reach_size.end(), v_point_reach_offset.begin());
    auto it = thrust::copy_if(v_point_reach_full.begin(), v_point_reach_full.end(), v_cell_reach.begin(), []__device__(int const &val) -> bool {
        return val >= 0;
    });
    v_cell_reach.resize(thrust::distance(v_cell_reach.begin(), it));
#ifdef DEBUG_ON
    std::cout << "v_cell_reach: " << v_cell_reach.size() << std::endl;
#endif
}

void nc_tree::process_points(d_vec<int> &v_point_id, d_vec<float> &v_point_data, magmaMPI mpi) noexcept {
    auto const it_status = v_coord_status.begin();
    thrust::for_each(v_point_id.begin(), v_point_id.end(), [=]__device__(int const &id) -> void {
        if (id >= 0) {
            // PROCESSED
            *(it_status + id) = 1;
        }
    });
    d_vec<int> v_point_index(v_point_id.size());
    index_points(v_point_data, v_point_index);
    // obtain reach
    d_vec<int> v_point_cells_in_reach;
    d_vec<int> v_point_cell_reach_offset(v_point_id.size());
    d_vec<int> v_point_cell_reach_size(v_point_id.size());

    collect_cells_in_reach(v_point_index, v_point_cells_in_reach, v_point_cell_reach_offset, v_point_cell_reach_size);

    d_vec<int> v_points_in_reach_size(v_point_id.size());
    d_vec<int> v_points_in_reach_offset(v_point_id.size());
    thrust::counting_iterator<int> it_point_begin(0);
    thrust::counting_iterator<int> it_point_end = it_point_begin + v_point_id.size();

    auto const it_coord_cell_size = v_coord_cell_size.begin();
    auto const it_point_cell_reach = v_point_cell_reach_size.begin();
    auto const it_point_cells_in_reach = v_point_cells_in_reach.begin();
    auto const it_point_cell_reach_offset = v_point_cell_reach_offset.begin();

    thrust::transform(it_point_begin, it_point_end, v_points_in_reach_size.begin(), [=]__device__(int const &i) -> int {
        auto p_sum = 0;
        for (int j = 0; j < *(it_point_cell_reach + i); ++j) {
            p_sum += *(it_coord_cell_size + *(it_point_cells_in_reach + *(it_point_cell_reach_offset + i) + j));
        }
        return p_sum;
    });
    thrust::exclusive_scan(v_points_in_reach_size.begin(), v_points_in_reach_size.end(), v_points_in_reach_offset.begin());
    auto table_size = thrust::reduce(v_points_in_reach_size.begin(), v_points_in_reach_size.end(), 0);
    std::cout << "table_size: " << table_size << std::endl;
    d_vec<int> v_hit_table_id_1(table_size, -1);
    d_vec<int> v_hit_table_id_2(table_size, -1);
    auto const it_hit_table_1 = v_hit_table_id_1.begin();
    auto const it_hit_table_2 = v_hit_table_id_2.begin();
    auto const it_points_in_reach_offset = v_points_in_reach_offset.begin();
    auto const it_points_in_reach_size = v_points_in_reach_size.begin();
    auto const it_coord_id = v_coord_id.begin();
    auto const it_coord_offset = v_coord_cell_offset.begin();
    thrust::for_each(it_point_begin, it_point_end, [=]__device__(int const &i) -> void {
        for (int j = 0; j < *(it_points_in_reach_size + i); ++j) {
            *(it_hit_table_1 + *(it_points_in_reach_offset + i) + j) = i;
        }
    });

    d_vec<int> v_cell_reach_size(v_point_cells_in_reach.size());
    d_vec<int> v_cell_reach_offset(v_point_cells_in_reach.size());
    thrust::transform(v_point_cells_in_reach.begin(), v_point_cells_in_reach.end(), v_cell_reach_size.begin(), [=]__device__(int const &c_id) -> int {
        return *(it_coord_cell_size + c_id);
    });
    thrust::exclusive_scan(v_cell_reach_size.begin(), v_cell_reach_size.end(), v_cell_reach_offset.begin());
    thrust::counting_iterator<int> it_cell_begin(0);
    thrust::counting_iterator<int> it_cell_end = it_cell_begin + v_point_cells_in_reach.size();
    auto const it_cell_reach_size = v_cell_reach_size.begin();
    auto const it_cell_reach_offset = v_cell_reach_offset.begin();
    thrust::for_each(it_cell_begin, it_cell_end, [=]__device__(int const &i) -> void {
        for (int j = 0; j < *(it_cell_reach_size + i); ++j) {
            *(it_hit_table_2 + *(it_cell_reach_offset + i) + j) = *(it_coord_id + *(it_coord_offset + *(it_point_cells_in_reach + i)) + j);
        }
    });

//    auto hit1 = thrust::reduce(v_hit_table_id_1.begin(), v_hit_table_id_1.end(), 0);
//    auto hit2 = thrust::reduce(v_hit_table_id_2.begin(), v_hit_table_id_2.end(), 0);
//    std::cout << "hit1: " << hit1 << " hit2: " << hit2 << std::endl;

    auto const it_coord_data = v_coord.begin();
    auto const it_point_data = v_point_data.begin();
    thrust::counting_iterator<int> it_table_cnt_begin(0);
    thrust::counting_iterator<int> it_table_cnt_end = it_table_cnt_begin + v_hit_table_id_1.size();
    // TODO REMOVE ?
//    auto it_perm_begin_1 = thrust::make_permutation_iterator(v_point_data.begin(), it_table_cnt_begin);
//    auto it_trans_begin_1 = thrust::make_transform_iterator(it_perm_begin_1, (thrust::placeholders::_1 * n_dim));

    float const _e2 = e2;
    float const _n_dim = n_dim;
    d_vec<float> v_result_table(table_size, 0);
    thrust::for_each(it_table_cnt_begin, it_table_cnt_end, [=]__device__(int const &i) -> void {
        auto it_point = it_point_data + (*(it_hit_table_1 + i) * _n_dim);
        auto it_coord = it_coord_data + (*(it_hit_table_2 + i) * _n_dim);
        float result = 0;
        #pragma unroll
        for (int d = 0; d < _n_dim; ++d) {
            result += (*(it_point + d) - *(it_coord + d)) * (*(it_point + d) - *(it_coord + d));
        }
        if (result > _e2) {
            *(it_hit_table_1 + i) = -1;
        }
    });

    d_vec<int> v_point_nn(v_point_id.size(), 0);
    auto const it_point_id = v_point_id.begin();
    auto const it_coord_nn = v_coord_nn.begin();
    thrust::transform(it_point_begin, it_point_end, v_point_nn.begin(), [=]__device__(int const &i) -> int {
        int p_m = 0;
        for (int j = 0; j < *(it_points_in_reach_size + i); ++j) {
            if (*(it_hit_table_1 + *(it_points_in_reach_offset + i) + j) != -1) {
                ++p_m;
            }
        }
        if (*(it_point_id + i) >= 0) {
            *(it_coord_nn + *(it_point_id + i)) = p_m;
        }
        return p_m;
    });

#ifdef MPI_ON
    mpi.allReduce(v_point_nn, magmaMPI::sum);
//    mpi.allReduce(v_coord_nn, magmaMPI::sum);
#endif


//    auto it_perm_begin = thrust::make_permutation_iterator(v_point_data.begin(), it_point_begin);
//    auto it_trans_begin_1 = thrust::make_transform_iterator(it_perm_begin_1, (thrust::placeholders::_1 * n_dim));
    auto it_point_nn = v_point_nn.begin();
    auto const _m = m;
    thrust::for_each(it_point_begin, it_point_end, [=]__device__(int const &i) -> void {
        if (*(it_point_nn + i) >= _m && *(it_point_id + i) >= 0) {
            *(it_coord_nn + *(it_point_id + i)) = *(it_point_nn + i);
//            *(it_coord_nn + *(it_point_id + i)) = _m;
        }
    });

//    d_vec<int> v_point_cluster(v_point_id.size(), NO_CLUSTER);
//    thrust::for_each(it_point_begin, it_point_end, [=]__device__(int const &i) -> void {

//    }
//    s_vec<int> v_point_new_cluster_mark(v_point_id.size(), 0);
//    s_vec<int> v_point_new_cluster_offset(v_point_id.size());

    /*
    exa::for_each(v_point_iota, 0, v_point_iota.size(), [&](int const &i) -> void {
        if (v_point_nn[i] >= m) {
            v_point_cluster[i] = i + cluster_size;
            if (v_point_id[i] >= 0) {
                v_coord_nn[v_point_id[i]] = v_point_nn[i];
                if (v_coord_cluster[v_point_id[i]] == NO_CLUSTER) {
                    v_coord_cluster[v_point_id[i]] = v_point_cluster[i];
                } else {
                    v_point_cluster[i] = v_coord_cluster[v_point_id[i]];
//                    std::cout << "CHECKPOINT" << std::endl;
                }
            }
        }
    });
     */

    print_cuda_memory_usage();

    /*
    bool is_done = false;
    int iter_cnt = 0;
    while (!is_done) {
        is_done = true;
        ++iter_cnt;
        exa::for_each(v_point_iota, 0, v_point_iota.size(), [&](int const &i) -> void {
            if (v_point_nn[i] >= m) {
                for (int j = 0; j < v_points_in_reach_size[i]; ++j) {
                    auto id2 = v_hit_table_id_2[v_points_in_reach_offset[i] + j];
                    if (id2 == -1) continue;
                    if (v_coord_nn[id2] >= m) {
                        if (v_coord_cluster[id2] == NO_CLUSTER) {
                            v_coord_cluster[v_point_id[i]] = v_point_cluster[i];
                        } else if (v_coord_cluster[id2] < v_point_cluster[i]) {
//                            if (v_point_cluster[i] != i + cluster_size) {
//                                std::cout << "CHECKPOINT!" << std::endl;
//                            }
                            v_point_cluster[i] = v_coord_cluster[id2];
                            if (v_point_id[i] >= 0) {
                                v_coord_cluster[v_point_id[i]] = v_point_cluster[i];
                            }
                            is_done = false;
                        }
                    }
                }
            }
        });
    }
    std::cout << "label iterations: " << iter_cnt << std::endl;
    int new_clusters = 0;
    exa::for_each(v_point_iota, 0, v_point_iota.size(), [&](int const &i) -> void {
        if (v_point_nn[i] >= m && v_point_cluster[i] == i + cluster_size) {
            ++new_clusters;
        }
    });
    std::cout << "new clusters: " << new_clusters << std::endl;
    cluster_size += new_clusters;


    exa::for_each(v_point_iota, 0, v_point_iota.size(), [&](int const &i) -> void {
        if (v_point_nn[i] >= m) {
//            assert(v_point_cluster[i] != NO_CLUSTER);
            for (int j = 0; j < v_points_in_reach_size[i]; ++j) {
                auto id2 = v_hit_table_id_2[v_points_in_reach_offset[i] + j];
                if (id2 == -1) continue;
                if (v_coord_cluster[id2] == NO_CLUSTER) {
                    v_coord_cluster[id2] = v_point_cluster[i];
                }
                else if (v_coord_cluster[id2] != v_point_cluster[i] && v_coord_nn[id2] >= m) {
                    std::cout << "CHECKPINT!!" << std::endl;
//                    assert(v_point_cluster[i] < v_coord_cluster[id2]);
//                    v_coord_cluster[id2] = v_point_cluster[i];
                }
            }
        }
    });
    */
}

void nc_tree::select_and_process(magmaMPI mpi) noexcept {
    v_coord_nn.resize(n_coord, 0);
    v_coord_cluster.resize(n_coord, NO_CLUSTER);
    v_coord_status.resize(n_coord, NOT_PROCESSED);

    d_vec<int> v_point_id(v_coord_id.size());
    thrust::sequence(v_point_id.begin(), v_point_id.end(), 0);

    d_vec<int> v_id_chunk;
    d_vec<float> v_data_chunk;
    int n_blocks = 1;
    for (int i = 0; i < 1/*n_blocks*/; ++i) {
        int block_size = magma_util::get_block_size(i, static_cast<int>(v_point_id.size()), n_blocks);
        int block_offset = magma_util::get_block_offset(i, static_cast<int>(v_point_id.size()), n_blocks);
        std::cout << "block offset: " << block_offset << " size: " << block_size << std::endl;
        v_id_chunk.clear();
        v_id_chunk.insert(v_id_chunk.begin(), v_point_id.begin() + block_offset, v_point_id.begin() + block_offset + block_size);
        v_data_chunk.clear();
        v_data_chunk.insert(v_data_chunk.begin(), v_coord.begin() + (block_offset * n_dim), v_coord.begin()
            + ((block_offset + block_size) * n_dim));
        process_points(v_id_chunk, v_data_chunk, mpi);
    }

}

void nc_tree::get_result_meta(int &cores, int &noise, int &clusters, int &n, magmaMPI mpi) noexcept {
    n = n_coord;

    auto const _m = m;
    cores = thrust::count_if(v_coord_nn.begin(), v_coord_nn.end(), [=]__device__(int const &v) -> bool {
        return v >= _m;
    });
    noise = thrust::count_if(v_coord_cluster.begin(), v_coord_cluster.end(), []__device__(int const &v) -> bool {
        return v >= 0;
    });

#ifdef MPI_ON
    d_vec<int> v_data(2);
    v_data[0] = cores;
    v_data[1] = noise;
    mpi.allReduce(v_data, magmaMPI::sum);
    cores = v_data[0];
    noise = v_data[1];
#endif

//    int cluster_points = 0;
//    for (auto const &cluster : v_coord_cluster) {
//        if (cluster >= 0) ++cluster_points;
//    }
//    noise = n_coord - cluster_points;

    /*
    std::unordered_map<int, int> v_cluster_map;
    for (int const &cluster : v_coord_cluster) {
        if (cluster >= 0) {
            auto elem = v_cluster_map.find(cluster);
            if (elem == v_cluster_map.end()) {
                v_cluster_map.insert(std::make_pair(cluster, 1));
            } else {
                (*elem).second++;
            }
        }
    }
    clusters = v_cluster_map.size();
     */
}

